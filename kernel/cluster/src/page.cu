#include "km_ops.h"
#include "pytorch_extension_utils.h"
#include "km_include/page.cuh"

using namespace flashinfer;

void append_kv_cache_prefill(torch::Tensor k,
							 torch::Tensor v,
							 torch::Tensor kv_data,
							 torch::Tensor kv_indices,
							 torch::Tensor kv_indptr,
							 unsigned int kv_last_page_len,
							 unsigned int kv_last_page_idx,
							 unsigned int layout) {
	constexpr size_t batch_size = 1;
	
#ifdef BSK_TORCH_CHECK
	CHECK_INPUT(k); // [bsz, num_kv_heads, head_dim]
	CHECK_INPUT(v); // [bsz, num_kv_heads, head_dim]
	// (num_max_pages, 2, H_kv, page_size, head_dim) for HND
	// (num_max_pages, 2, page_size, H_kv, head_dim) for NHD
	CHECK_INPUT(kv_data);
	CHECK_INPUT(kv_indices); // [num_pages]

	CHECK_DIM(1, kv_indices);
	CHECK_DIM(3, k);
	CHECK_DIM(3, v);
	CHECK_DIM(5, kv_data);

	CHECK_GE(k.size(0), 2); // Prefill
	CHECK_GE(v.size(0), 2); // Prefill
	CHECK_EQ(kv_indices.scalar_type(), torch::kInt32);
	CHECK_EQ(kv_indptr.scalar_type(), torch::kInt32);
#endif

	size_t seq_len = k.size(0);
	size_t num_kv_heads = k.size(1);
	size_t head_dim = k.size(2);
	size_t page_size;
	QKVLayout kv_layout = static_cast<QKVLayout>(layout);
	if(kv_layout == QKVLayout::kHND) {
		assert(false);
	} else {
		page_size = kv_data.size(2);
#ifdef BSK_TORCH_CHECK
		CHECK_EQ(kv_data.size(3), num_kv_heads);
		CHECK_EQ(kv_data.size(4), head_dim);
#endif
	}

#ifdef BSK_TORCH_CHECK
	CHECK_EQ(seq_len, v.size(0));
#endif

	torch::Tensor append_indptr =
		torch::tensor({0, static_cast<int32_t>(seq_len)}, kv_indices.options());

	bool success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(k.scalar_type(), c_type, [&] {
		SWITCH_LAYOUT(kv_layout, KV_LAYOUT, {
			paged_kv_t<PageStorage::kIndices, KV_LAYOUT, c_type, int32_t> paged_kv(
				num_kv_heads,
				page_size,
				head_dim,
				batch_size,
				0,
				kv_last_page_len,
				kv_last_page_idx,
				static_cast<c_type*>(kv_data.data_ptr()),
				static_cast<int32_t*>(kv_indices.data_ptr()),
				static_cast<int32_t*>(kv_indptr.data_ptr()));

			hipError_t status =
				AppendPagedKVCachePrefill<PageStorage::kIndices, KV_LAYOUT, c_type, int32_t>(
					paged_kv,
					static_cast<c_type*>(k.data_ptr()),
					static_cast<c_type*>(v.data_ptr()),
					static_cast<int32_t*>(append_indptr.data_ptr()),
					nullptr);

			TORCH_CHECK(status == hipSuccess,
						"Append_kv_cache_prefill failed with error code ",
						hipGetErrorString(status));
		});
		return true;
	});

	TORCH_CHECK(success, "Append_kv_cache_prefill failed to dispatch with dtype ", k.scalar_type());
}

void append_kv_cache_decode(torch::Tensor k,
							torch::Tensor v,
							torch::Tensor kv_data,
							torch::Tensor kv_indices,
							torch::Tensor kv_indptr,
							unsigned int kv_last_page_len,
							unsigned int kv_last_page_idx,
							unsigned int layout) {
	constexpr size_t batch_size = 1;

#ifdef BSK_TORCH_CHECK
	CHECK_INPUT(k); // [bsz, num_kv_heads, head_dim]
	CHECK_INPUT(v); // [bsz, num_kv_heads, head_dim]
	// (num_max_pages, 2, H_kv, page_size, head_dim) for HND
	// (num_max_pages, 2, page_size, H_kv, head_dim) for NHD
	CHECK_INPUT(kv_data);
	CHECK_INPUT(kv_indices); // [num_pages]

	CHECK_DIM(1, kv_indices);
	CHECK_DIM(3, k);
	CHECK_DIM(3, v);
	CHECK_DIM(5, kv_data);

	CHECK_EQ(k.size(0), 1); // decode
	CHECK_EQ(v.size(0), 1); // decode
	CHECK_EQ(kv_indices.scalar_type(), torch::kInt32);
	CHECK_EQ(kv_indptr.scalar_type(), torch::kInt32);
#endif

	size_t num_kv_heads = k.size(1);
	size_t head_dim = k.size(2);
	size_t page_size;
	QKVLayout kv_layout = static_cast<QKVLayout>(layout);
	if(kv_layout == QKVLayout::kHND) {
		assert(false);
	} else {
		page_size = kv_data.size(2);
		CHECK_EQ(kv_data.size(3), num_kv_heads);
		CHECK_EQ(kv_data.size(4), head_dim);
	}
	
	bool success = DISPATCH_PYTORCH_DTYPE_TO_CTYPE(k.scalar_type(), c_type, [&] {
		SWITCH_LAYOUT(kv_layout, KV_LAYOUT, {
			paged_kv_t<PageStorage::kIndices, KV_LAYOUT, c_type, int32_t> paged_kv(
				num_kv_heads,
				page_size,
				head_dim,
				batch_size,
				0,
				kv_last_page_len,
				kv_last_page_idx,
				static_cast<c_type*>(kv_data.data_ptr()),
				static_cast<int32_t*>(kv_indices.data_ptr()),
				static_cast<int32_t*>(kv_indptr.data_ptr()));

			hipError_t status =
				AppendPagedKVCacheDecode<PageStorage::kIndices, KV_LAYOUT, c_type, int32_t>(
					paged_kv,
					static_cast<c_type*>(k.data_ptr()),
					static_cast<c_type*>(v.data_ptr()),
					nullptr);

			TORCH_CHECK(status == hipSuccess,
						"Append_kv_cache_decode failed with error code ",
						hipGetErrorString(status));
		});
		return true;
	});

	TORCH_CHECK(success, "Append_kv_cache_decode failed to dispatch with dtype ", k.scalar_type());
}